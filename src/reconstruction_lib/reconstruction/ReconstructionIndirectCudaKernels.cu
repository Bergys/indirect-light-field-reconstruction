#include "hip/hip_runtime.h"
//-------------------------------------------------------------------------------------------------
// CUDA reconstruction kernels
//-------------------------------------------------------------------------------------------------

#include "ReconstructionIndirectCudaKernels.hpp"
#include <stdio.h>

enum
{
	STACK_SIZE          = 64,
	MAX_SAMPLES         = 512,
	SMALL_SURFACE_LIMIT = 4,
	SENTINEL            = 0x76543210,
};

#define HULL_TYPE Vec4f
#define HULL_EMPTY (Vec4f(+FW_F32_MAX, -FW_F32_MAX, +FW_F32_MAX, -FW_F32_MAX))
#define HULL_ADD(abox,v) do {\
	float dx = (v).x;\
	float dy = (v).y;\
	float k = dx/dy;\
	if (dy >= 0.f)	abox.x = fmin(abox.x, k), abox.y = fmax(abox.y, k);\
	else			abox.z = fmin(abox.z, k), abox.w = fmax(abox.w, k);\
	} while (0)
#define HULL_INSIDE(abox) (abox.y > abox.z && abox.x < abox.w)

using namespace FW;

//------------------------------------------------------------------------

__device__ __inline__ U32   getLo                   (U64 a)                 { return __double2loint(__longlong_as_double(a)); }
__device__ __inline__ S32   getLo                   (S64 a)                 { return __double2loint(__longlong_as_double(a)); }
__device__ __inline__ U32   getHi                   (U64 a)                 { return __double2hiint(__longlong_as_double(a)); }
__device__ __inline__ S32   getHi                   (S64 a)                 { return __double2hiint(__longlong_as_double(a)); }
__device__ __inline__ U64   combineLoHi             (U32 lo, U32 hi)        { return __double_as_longlong(__hiloint2double(hi, lo)); }
__device__ __inline__ S64   combineLoHi             (S32 lo, S32 hi)        { return __double_as_longlong(__hiloint2double(hi, lo)); }
__device__ __inline__ U32   getLaneMaskLt           (void)                  { U32 r; asm("mov.u32 %0, %lanemask_lt;" : "=r"(r)); return r; }
__device__ __inline__ U32   getLaneMaskLe           (void)                  { U32 r; asm("mov.u32 %0, %lanemask_le;" : "=r"(r)); return r; }
__device__ __inline__ U32   getLaneMaskGt           (void)                  { U32 r; asm("mov.u32 %0, %lanemask_gt;" : "=r"(r)); return r; }
__device__ __inline__ U32   getLaneMaskGe           (void)                  { U32 r; asm("mov.u32 %0, %lanemask_ge;" : "=r"(r)); return r; }
__device__ __inline__ int   findLeadingOne          (U32 v)                 { U32 r; asm("bfind.u32 %0, %1;" : "=r"(r) : "r"(v)); return r; }
__device__ __inline__ int   findLastOne             (U32 v)                 { return 31-findLeadingOne(__brev(v)); } // 0..31, 32 for not found
__device__ __inline__ bool  singleLane              (void)                  { return ((__ballot(true) & getLaneMaskLt()) == 0); }
__device__ __inline__ U32   rol						(U32 x, U32 s)			{ return (x<<s)|(x>>(32-s)); }
__device__ __inline__ Vec2f U64toVec2f				(U64 xy)				{ return Vec2f(__int_as_float(getLo(xy)), __int_as_float(getHi(xy))); }
__device__ __inline__ U64   Vec2ftoU64				(const Vec2f& v)		{ return combineLoHi(__float_as_int(v.x), __float_as_int(v.y)); }
__device__ __inline__ int   imin					(S32 a, S32 b)			{ S32 v; asm("min.s32 %0, %1, %2;" : "=r"(v) : "r"(a), "r"(b)); return v; }
__device__ __inline__ int   imax					(S32 a, S32 b)			{ S32 v; asm("max.s32 %0, %1, %2;" : "=r"(v) : "r"(a), "r"(b)); return v; }

//------------------------------------------------------------------------

#define FW_HASH_MAGIC   (0x9e3779b9u)
#define FW_JENKINS_MIX(a, b, c)   \
    a -= b; a -= c; a ^= (c>>13); \
    b -= c; b -= a; b ^= (a<<8);  \
    c -= a; c -= b; c ^= (b>>13); \
    a -= b; a -= c; a ^= (c>>12); \
    b -= c; b -= a; b ^= (a<<16); \
    c -= a; c -= b; c ^= (b>>5);  \
    a -= b; a -= c; a ^= (c>>3);  \
    b -= c; b -= a; b ^= (a<<10); \
    c -= a; c -= b; c ^= (b>>15);

__device__ __inline__ U32  hashBits(U32 a, U32 b = FW_HASH_MAGIC, U32 c = 0)
{
	c += FW_HASH_MAGIC;
	FW_JENKINS_MIX(a, b, c);
	return c;
}

//------------------------------------------------------------------------

__device__ Mat3f orthogonalBasis(const Vec3f& v)
{
	Mat3f m;
	Vec3f mx = v;
	if( fabs( mx.x ) > fabs( mx.y ) && fabs( mx.x ) > fabs( mx.z ) )
	{
		FW::swap( mx.x, mx.y );
		mx.x = -mx.x;
	}
	else if( fabs( mx.y ) > fabs( mx.x ) && fabs( mx.y ) > fabs( mx.z ) )
	{
		FW::swap( mx.y, mx.z );
		mx.y = -mx.y;
	}
	else 
	{
		FW::swap( mx.z, mx.x );
		mx.z = -mx.z;
	}
	m.setCol( 1, cross(v, mx).normalized() );
	m.setCol( 0, cross(m.getCol(1), v).normalized() );
	m.setCol( 2, v );
	return m;
}

__device__ Vec2f toUnitDisk(const Vec2f& onSquare) 
{
	const float PI = 3.1415926535897932384626433832795f;
	float phi, r, u, v;
	float a = 2.f * onSquare.x - 1.f;
	float b = 2.f * onSquare.y - 1.f;

	if (a > -b)
	{
		if (a > b)
		{
			r=a;
			phi = (PI/4 ) * (b/a);
		}
		else
		{
			r = b;
			phi = (PI/4) * (2 - (a/b));
		}
	}
	else
	{
		if (a < b)
		{
			r = -a;
			phi = (PI/4) * (4 + (b/a));
		}
		else
		{
			r = -b;
			if (b != 0)	phi = (PI/4) * (6 - (a/b));
			else		phi = 0;
		}
	}
	u = r * (float)cosf(phi);
	v = r * (float)sinf(phi);
	return Vec2f(u, v);
}

__device__ Vec3f diskToCosineHemisphere(const Vec2f& disk)
{
	return Vec3f(disk.x, disk.y, sqrtf(fabsf(1.f - dot(disk, disk))));
}

__device__ Vec3f squareToCosineHemisphere(const Vec2f& square)
{
	return diskToCosineHemisphere(toUnitDisk(square));
}

__device__ float fminf(float a, float b, float c)				{ return fminf(fminf(a, b), c); }
__device__ float fminf(float a, float b, float c, float d)		{ return fminf(fminf(fminf(a, b), c), d); }
__device__ float fmaxf(float a, float b, float c)				{ return fmaxf(fmaxf(a, b), c); }
__device__ float fmaxf(float a, float b, float c, float d)		{ return fmaxf(fmaxf(fmaxf(a, b), c), d); }

__device__ 	bool insideConvexHull(U64* samples, int lo, int hi, Vec3f& origin, Mat3f& basis)
{
	if (hi - lo > 3)
		return true;

	float Rscale;
	switch(hi - lo)
	{
		case 1:	Rscale = 0.5f; break;
		case 2:	Rscale = 0.6f; break;
		case 3:	Rscale = 0.7f; break;
	}

	HULL_TYPE hull = HULL_EMPTY;
	for (int i=lo; i < hi; i++)	
	{
		U64 st = samples[MAX_SAMPLES - i - 1];

#if 1
		int sidx = getLo(st) * 2;
		float4 td0 = tex1Dfetch(t_samples, sidx + 0);
		float4 td1 = tex1Dfetch(t_samples, sidx + 1);
		Vec3f p = Vec3f(td0.x, td0.y, td0.z) - origin;
		Vec3f n = Vec3f(td1.x, td1.y, td1.z);
		float R = fabsf(td0.w);
#else
		CudaSampleInd& s = ((CudaSampleInd*)in.samples)[getLo(st)];
		Vec3f p = s.pos - origin;
		Vec3f n = s.normal;
		float R = fabsf(s.size);
#endif

		Vec3f P = basis * p; // splat center in camera space
		Vec3f N = basis * n; // splat normal in camera space
		float invw = 1.f / P.z;
		R *= invw;

		const float cosAngle = fabsf(N.z);
		Vec2f xy  = P.getXY() * invw;
		Vec2f N2d = N.getXY().normalized();
		if((__float_as_int(N2d.x) | __float_as_int(N2d.y)) == 0)
			N2d.x = 1.f;

		const float minorScale = Rscale * R * cosAngle;
		const float majorScale = Rscale * R;
		const Vec2f minorAxis  = minorScale * Vec2f(N2d.x,  N2d.y);
		const Vec2f majorAxis  = majorScale * Vec2f(N2d.y, -N2d.x);

		HULL_ADD(hull, xy + minorAxis);
		HULL_ADD(hull, xy - minorAxis);
		HULL_ADD(hull, xy + majorAxis);
		HULL_ADD(hull, xy - majorAxis);
		
		const float diagScale = .5f * Rscale * R * sqrtf(1 + cosAngle*cosAngle);
		const Vec2f diag1Axis = diagScale * Vec2f(N2d.x + N2d.y, N2d.y - N2d.x);
		const Vec2f diag2Axis = diagScale * Vec2f(N2d.x - N2d.y, N2d.y + N2d.x);

		HULL_ADD(hull, xy + diag1Axis);
		HULL_ADD(hull, xy - diag1Axis);
		HULL_ADD(hull, xy + diag2Axis);
		HULL_ADD(hull, xy - diag2Axis);
		
		if (HULL_INSIDE(hull))
			return true;
	}

	return false;
}

//------------------------------------------------------------------------

#define storeResult(idx_,color_) do {\
	float* cptr = (float*)(in.resultImg + ((idx_) << 4)); \
	atomicAdd(&cptr[0], (color_).x); \
	atomicAdd(&cptr[1], (color_).y); \
	atomicAdd(&cptr[2], (color_).z); \
	atomicAdd(&cptr[3], (color_).w); \
} while (0)

__device__ void accumulateColor(Vec4f& color, const Vec4f& c)
{
#ifdef SELECT_NEAREST_SAMPLE
	if (c.w > color.w)
		color = c;
#else
	color += c;
#endif
}

__device__ float vMFfromBandwidth(float bw)
{
	return 4.0f * sqrtf(bw);
}

//------------------------------------------------------------------------

extern "C" __global__ void __launch_bounds__(128,6) filterKernel(void)
{
    const IndirectKernelInput& in = *(const IndirectKernelInput*)&c_IndirectKernelInput;

    int tidx = threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * (blockIdx.x + gridDim.x * blockIdx.y));
	if (tidx >= in.numRays)
		return;

	const float t_eps   = 1e-3f;
	const float ray_eps = 1e-20f;

	int reconRayIdx = tidx + in.firstRay;
	int receiverIdx = reconRayIdx / in.nr;
	int rayPixelIdx = reconRayIdx % in.outputSpp;

	int   pixel;
	Vec3f origin;
	Vec3f direction;
	Vec3f weight;

	if (in.pbrtRayCount > 0)
	{
		CudaPBRTRay* ray = &((CudaPBRTRay*)in.pbrtRays)[tidx];

		pixel     = ray->pixel;
		origin    = ray->o;
		direction = ray->d;
		weight    = ray->weight;

		if (direction.isZero() || weight.isZero())
		{
			Vec4f black(0,0,0,1);
			storeResult(pixel, black);
			return;
		}
	} else
	{
		// construct reconstruction ray

		CudaReceiverInd* recv = &((CudaReceiverInd*)in.recv)[receiverIdx];

		pixel = recv->pixel;
		Vec2f dsqr = ((Vec2f*)in.sobol)[rayPixelIdx];
		dsqr.x += hashBits(hashBits(pixel, FW_HASH_MAGIC, 0)) * (1.f / FW_U32_MAX);
		dsqr.y += hashBits(hashBits(pixel, FW_HASH_MAGIC, 1)) * (1.f / FW_U32_MAX);
		if (dsqr.x >= 1.f) dsqr.x -= 1.f;
		if (dsqr.y >= 1.f) dsqr.y -= 1.f;
		Vec3f dunit     = squareToCosineHemisphere(dsqr);

		origin    = recv->pos + recv->normal * t_eps;
		direction = (orthogonalBasis(recv->normal) * dunit).normalized();
		weight    = recv->albedo;
	}

	Mat3f basis = orthogonalBasis(direction).transposed();

	// traversal stack
	int stack[STACK_SIZE];
	stack[0] = SENTINEL; // Bottom-most entry.
	char* stackPtr = (char*)&stack[0];

	// sample array
	U64 samples[MAX_SAMPLES];	// lo = idx, hi = key
	int numSamples = 0;

	// ray parameters
	Vec3f idir;
	idir.x = 1.0f / (fabs(direction.x) > ray_eps ? direction.x : (direction.x < 0 ? -ray_eps : ray_eps));
	idir.y = 1.0f / (fabs(direction.y) > ray_eps ? direction.y : (direction.y < 0 ? -ray_eps : ray_eps));
	idir.z = 1.0f / (fabs(direction.z) > ray_eps ? direction.z : (direction.z < 0 ? -ray_eps : ray_eps));
	Vec3f ood = origin * idir;

	// todo: heapify?

	int nodeAddr = 0;
	int leafAddr = 0;
	while(nodeAddr != SENTINEL)
	{
		bool searchingLeaf = true;
		while (nodeAddr >= 0 && nodeAddr != SENTINEL)
		{
			// internal node, intersect against child nodes
			int idx = nodeAddr;
#if 1
			float4 hdr  = tex1Dfetch(t_nodes, idx*4);
			float4 n0xy = tex1Dfetch(t_nodes, idx*4+1);
			float4 n1xy = tex1Dfetch(t_nodes, idx*4+2);
			float4 nz   = tex1Dfetch(t_nodes, idx*4+3);
			int idx0 = __float_as_int(hdr.x);
			int idx1 = __float_as_int(hdr.y);
#else
			CudaNodeInd& node = ((CudaNodeInd*)in.nodes)[idx];
			int idx0 = node.idx0;
			int idx1 = node.idx1;
			float4 n0xy = make_float4(node.bbmin[0].x, node.bbmax[0].x, node.bbmin[0].y, node.bbmax[0].y);
			float4 n1xy = make_float4(node.bbmin[1].x, node.bbmax[1].x, node.bbmin[1].y, node.bbmax[1].y);
			float4 nz   = make_float4(node.bbmin[0].z, node.bbmax[0].z, node.bbmin[1].z, node.bbmax[1].z);
#endif
			float c0lox = n0xy.x * idir.x - ood.x;
			float c0hix = n0xy.y * idir.x - ood.x;
			float c0loy = n0xy.z * idir.y - ood.y;
			float c0hiy = n0xy.w * idir.y - ood.y;
			float c0loz = nz.x   * idir.z - ood.z;
			float c0hiz = nz.y   * idir.z - ood.z;
			float c1loz = nz.z   * idir.z - ood.z;
			float c1hiz = nz.w   * idir.z - ood.z;
			float c0min = fmaxf(fminf(c0lox, c0hix), fminf(c0loy, c0hiy), fminf(c0loz, c0hiz), 0.f);
			float c0max = fminf(fmaxf(c0lox, c0hix), fmaxf(c0loy, c0hiy), fmaxf(c0loz, c0hiz));
			float c1lox = n1xy.x * idir.x - ood.x;
			float c1hix = n1xy.y * idir.x - ood.x;
			float c1loy = n1xy.z * idir.y - ood.y;
			float c1hiy = n1xy.w * idir.y - ood.y;
			float c1min = fmaxf(fminf(c1lox, c1hix), fminf(c1loy, c1hiy), fminf(c1loz, c1hiz), 0.f);
			float c1max = fminf(fmaxf(c1lox, c1hix), fmaxf(c1loy, c1hiy), fmaxf(c1loz, c1hiz));

			bool traverseChild0 = (c0max >= c0min);
			bool traverseChild1 = (c1max >= c1min);

			if (!traverseChild0 && !traverseChild1)
			{
				// Neither child was intersected => pop stack.
				nodeAddr = *(int*)stackPtr;
				stackPtr -= 4;
			}
			else
			{
				// Otherwise fetch child pointers.
				// todo postpone fetch until here
				nodeAddr = (traverseChild0) ? idx0 : idx1;

				// Both children were intersected => push one.
				if (traverseChild0 && traverseChild1)
				{
					if (c1min < c0min)
						swap(nodeAddr, idx1);
					stackPtr += 4;
					*(int*)stackPtr = idx1;
				}
			}

			// First leaf => postpone and continue traversal.
			if (nodeAddr < 0 && leafAddr >= 0)
			{
				searchingLeaf = false;
				leafAddr = nodeAddr;
				nodeAddr = *(int*)stackPtr;
				stackPtr -= 4;
			}

			// All SIMD lanes have found a leaf => process them.
			if (!__any(searchingLeaf))
				break;
		}

		// Process postponed leaf nodes.

		while (leafAddr < 0)
		{
			// leaf node, test against samples here
			float ssize = 0.f;
			for (int i = ~leafAddr; ssize >= 0.f; i++)
			{
#if 1
				int sidx = i * 2;
				float4 td0 = tex1Dfetch(t_samples, sidx+0);
				float4 td1 = tex1Dfetch(t_samples, sidx+1);
				Vec3f spos(td0.x, td0.y, td0.z);
				Vec3f snormal(td1.x, td1.y, td1.z);
				float splen = td1.w;
				ssize = td0.w;
#else
				CudaSampleInd& s = ((CudaSampleInd*)in.samples)[i];
				Vec3f spos = s.pos;
				Vec3f snormal = s.normal;
				float splen = s.plen;
				ssize = s.size;
#endif
				// distance to splat plane along ray direction
				Vec3f y  = spos - origin;
				float yn = dot(y, snormal);			// if positive, splat is back-facing
				float dn = dot(direction, snormal);	// if positive, splat is back-facing
				float t = yn * (1.f / dn);
				if (t < t_eps * splen) // epsilon based on ray length
					continue; // avoid hitting the originating surface

				// ray intersection on splat plane
				Vec3f p = origin + t * direction;
				if ((p - spos).lenSqr() > ssize * ssize)
					continue; // does not hit the splat

				// distance to splat center from ray normal plane
				float dist_st = dot(y, direction);

				// back-face in the nearfield -> cull (basically because small concavities suffer from undersampling)
				if (yn >= 0.f && dist_st < fabsf(ssize))
					continue;

				// rare but possible
				if (numSamples == MAX_SAMPLES)
				{
					atomicAdd(&g_overflowCount, 1);
					return;
				}

				// add sample into a binary heap
				int x = numSamples;
				while (x > 0)
				{
					int   p = ((x+1) >> 1) - 1;
					U64   sp = samples[p];
					float tp = __int_as_float(getHi(sp));
					if (tp > t)
						samples[x] = sp;
					else
						break;
					x = p;
				}
				samples[x] = combineLoHi(i, __float_as_int(t));
				numSamples++;
			}

			// Another leaf was postponed => process it as well.

			leafAddr = nodeAddr;
			if(nodeAddr < 0)
			{
				nodeAddr = *(int*)stackPtr;
				stackPtr -= 4;
			}
		} // leaf
	} // traversal

	// no samples? terminate
	if (!numSamples)
	{
		atomicAdd(&g_emptyCount, 1);
		return;
	}

	// process in near->far order
	Vec4f color = 0.f;
	int heapsize = numSamples;
	int first      = 0; // first sample of the current surface
	int firstaccum = 0; // first sample we have accumulated so far
	int idx        = 0;
	for (; idx < numSamples; idx++)
	{
		// extract top of heap
		U64 st = samples[0];

		// pop last and restore heap property
		heapsize--;
		if (heapsize)
		{
			U64   sb = samples[heapsize];
			float tb = __int_as_float(getHi(sb));
			int   x  = 0;
			int   c0 = ((x+1)*2)-1;
			int   c1 = c0 + 1;
			while (c0 < heapsize)
			{
				U64   sc0 = samples[c0];
				U64   sc1 = c1 < heapsize ? samples[c1] : combineLoHi(0, __float_as_int(FW_F32_MAX));
				float tc0 = __int_as_float(getHi(sc0));
				float tc1 = __int_as_float(getHi(sc1));
				if (fminf(tc0, tc1) < tb)
				{
					bool min0 = (tc0 < tc1);
					samples[x] = min0 ? sc0 : sc1;
					x = min0 ? c0 : c1;
					c0 = 2*x + 1;
					c1 = c0 + 1;
				} else
					break;
			}
			samples[x] = sb;
		}

		// store sample
		samples[MAX_SAMPLES - idx - 1] = st;

		// get sample data
#if 0
		// debug debug this is broken! no color
		int            sidx   = getLo(st) * 2;
		float          t      = __int_as_float(getHi(st));
		float4         td0    = tex1Dfetch(t_samples, sidx+0);
		float4         td1    = tex1Dfetch(t_samples, sidx+1);
		float4         td2    = tex1Dfetch(t_samples, sidx+2);
		Vec3f          n      (td1.x, td1.y, td1.z);
		Vec3f          spos   (td0.x, td0.y, td0.z);
		float          ssize  = fabsf(td0.w);
		Vec3f          scolor (td2.x, td2.y, td2.z);
#else
		CudaSampleInd& s      = ((CudaSampleInd*)in.samples)[getLo(st)];
		float          t      = __int_as_float(getHi(st));
		Vec3f          n      = s.normal;
		Vec3f          spos   = s.pos;
		float          ssize  = fabsf(s.size);
		Vec3f          scolor = s.color;
#endif

		Vec4f c(0.f); // color to be accumulated
		if (dot(n, direction) >= 0)
		{
			n = -n;
		} else
		{
			// calculate color to be accumulated
			#ifdef AMBIENT_OCCLUSION
			{
				c = (t < AMBIENT_OCCLUSION) ? 0.f : 1.f;
				c.w = 1.f;
			}
			#else
			{
				Vec3f p = origin + t * direction;

				#ifdef USE_BANDWIDTH_INFORMATION
				{
					float anglecos = dot((s.orig - spos).normalized(), -direction);
					float bw = vMFfromBandwidth(s.bw);
					float vMF = expf(bw * anglecos - bw); // vMF but normalized to [0,1]

					// How large is the splat compared to the length of the ray?
#if 0
					float distWeight = fmin(1.f, ssize / t);
					const float spatialWeight = fmax(0.0f, 1.0f - (p - spos).length() / ssize);
					float w = vMF * fmin(1.f, spatialWeight / distWeight);
#else
					// another way: distance on UV plane between rec.ray and ray to splat center (equally good)
					float d = 1.f / dot(direction, (spos - origin).normalized());
					d = .5f * sqrtf(fabsf(d*d - 1.f)); // times 0.5 for good measure
					float w = vMF * fmax(0.f, 1.f - d);
#endif
					c = Vec4f(w * scolor, w);
				}
				#else
				{
					float w = 1.f - (spos - p).length() / ssize;
					c = Vec4f(w * scolor, w);
				}
				#endif
			}
			#endif
		}

		// test against previous samples in surface
		bool conflict = false;
		for (int i=first; i < idx && !conflict; i++)
		{
			U64 st0 = samples[MAX_SAMPLES - i - 1];
			CudaSampleInd& s0 = ((CudaSampleInd*)in.samples)[getLo(st0)];
			float          t0 = __int_as_float(getHi(st0));
			Vec3f          n0 = s0.normal;
			if (dot(n0, direction) >= 0)
				n0 = -n0;

			// consistently facing (away from) each other?
			Vec3f diff = (spos - s0.pos);
			float invlen = rsqrtf(diff.lenSqr());
			float cosAngle1 =  dot(n0, diff);
			float cosAngle2 = -dot(n,  diff);
			const float eps = 0.035f;
			conflict = (fmaxf(fminf(cosAngle1, cosAngle2), -fmaxf(cosAngle1, cosAngle2)) * invlen < -eps);
		}

		// if there wasn't a conflict, accumulate and continue
		if (!conflict)
		{
			accumulateColor(color, c);
			continue;
		}

		// there was a conflict, always start a logically new surface
		first = idx;

		// if not enough samples, accumulate and continue
		if (idx - firstaccum < SMALL_SURFACE_LIMIT)
		{
			// but if the surface covers the sample in convex hull sense, use it!
			if (insideConvexHull(samples, firstaccum, idx, origin, basis))
				break;

			accumulateColor(color, c);
			continue;
		}

		// convex hull was okay, break and return result
		break;
	}

	// if nothing found, treat as black (e.g. only backfacing splats)
	if (color.w == 0.f)
		color.w = 1.f;

	// normalize color
	color *= 1.f/color.w;

#ifndef AMBIENT_OCCLUSION
	color.x *= weight.x;
	color.y *= weight.y;
	color.z *= weight.z;
#endif

	storeResult(pixel, color); 
}

//------------------------------------------------------------------------

extern "C" __global__ void __launch_bounds__(128,6) shrinkKernel(void)
{
    const IndirectKernelInput& in = *(const IndirectKernelInput*)&c_IndirectKernelInput;

    int tidx = threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * (blockIdx.x + gridDim.x * blockIdx.y));
	if (tidx >= in.numRays)
		return;
	tidx += in.firstRay;

	const float ray_eps = 1e-20f;

	// fetch ray
	CudaShrinkRayInd& ray = ((CudaShrinkRayInd*)in.rays)[tidx];
	Vec3f origin    = ray.origin;
	Vec3f hitp      = ray.endpoint;
	Vec3f direction = hitp - origin;
	float rayLen = direction.length();
	direction *= 1.f / rayLen; // normalize

	// traversal stack
	int stack[STACK_SIZE];
	stack[0] = SENTINEL; // Bottom-most entry.
	char* stackPtr = (char*)&stack[0];

	// ray traversal parameters
	Vec3f idir;
	idir.x = 1.0f / (fabs(direction.x) > ray_eps ? direction.x : (direction.x < 0 ? -ray_eps : ray_eps));
	idir.y = 1.0f / (fabs(direction.y) > ray_eps ? direction.y : (direction.y < 0 ? -ray_eps : ray_eps));
	idir.z = 1.0f / (fabs(direction.z) > ray_eps ? direction.z : (direction.z < 0 ? -ray_eps : ray_eps));
	Vec3f ood = origin * idir;

	// adaptive epsilon similar to from PBRT (takes max because for defocus the rays start from the origin of camera space)
	float eps = 1e-3f * fmaxf(origin.length(), hitp.length());

	int nodeAddr = 0;
	int leafAddr = 0;
	while(nodeAddr != SENTINEL)
	{
		bool searchingLeaf = true;
		while (nodeAddr >= 0 && nodeAddr != SENTINEL)
		{
			// internal node, intersect against child nodes
			int idx = nodeAddr;
			float4 hdr  = tex1Dfetch(t_nodes, idx*4);
			float4 n0xy = tex1Dfetch(t_nodes, idx*4+1);
			float4 n1xy = tex1Dfetch(t_nodes, idx*4+2);
			float4 nz   = tex1Dfetch(t_nodes, idx*4+3);
			int idx0 = __float_as_int(hdr.x);
			int idx1 = __float_as_int(hdr.y);
			float c0lox = n0xy.x * idir.x - ood.x;
			float c0hix = n0xy.y * idir.x - ood.x;
			float c0loy = n0xy.z * idir.y - ood.y;
			float c0hiy = n0xy.w * idir.y - ood.y;
			float c0loz = nz.x   * idir.z - ood.z;
			float c0hiz = nz.y   * idir.z - ood.z;
			float c1loz = nz.z   * idir.z - ood.z;
			float c1hiz = nz.w   * idir.z - ood.z;
			float c0min = fmaxf(fminf(c0lox, c0hix), fminf(c0loy, c0hiy), fminf(c0loz, c0hiz), 0.f);
			float c0max = fminf(fmaxf(c0lox, c0hix), fmaxf(c0loy, c0hiy), fmaxf(c0loz, c0hiz));
			float c1lox = n1xy.x * idir.x - ood.x;
			float c1hix = n1xy.y * idir.x - ood.x;
			float c1loy = n1xy.z * idir.y - ood.y;
			float c1hiy = n1xy.w * idir.y - ood.y;
			float c1min = fmaxf(fminf(c1lox, c1hix), fminf(c1loy, c1hiy), fminf(c1loz, c1hiz), 0.f);
			float c1max = fminf(fmaxf(c1lox, c1hix), fmaxf(c1loy, c1hiy), fmaxf(c1loz, c1hiz));

			bool traverseChild0 = (c0max >= c0min);
			bool traverseChild1 = (c1max >= c1min);

			if (!traverseChild0 && !traverseChild1)
			{
				// Neither child was intersected => pop stack.
				nodeAddr = *(int*)stackPtr;
				stackPtr -= 4;
			}
			else
			{
				// Otherwise fetch child pointers.
				// todo postpone fetch until here
				nodeAddr = (traverseChild0) ? idx0 : idx1;

				// Both children were intersected => push one.
				if (traverseChild0 && traverseChild1)
				{
					if (c1min < c0min)
						swap(nodeAddr, idx1);
					stackPtr += 4;
					*(int*)stackPtr = idx1;
				}
			}

			// First leaf => postpone and continue traversal.
			if (nodeAddr < 0 && leafAddr >= 0)
			{
				searchingLeaf = false;
				leafAddr = nodeAddr;
				nodeAddr = *(int*)stackPtr;
				stackPtr -= 4;
			}

			// All SIMD lanes have found a leaf => process them.
			if (!__any(searchingLeaf))
				break;
		}

		// Process postponed leaf nodes.

		while (leafAddr < 0)
		{
			// leaf node, test against samples here
			float ssize = 0.f;
			for (int i = ~leafAddr; ssize >= 0.f; i++)
			{
				int sidx = i * 2;
				float4 td0 = tex1Dfetch(t_samples, sidx+0);
				float4 td1 = tex1Dfetch(t_samples, sidx+1);
				Vec3f spos(td0.x, td0.y, td0.z);
				Vec3f snormal(td1.x, td1.y, td1.z);
				ssize = td0.w;

				// distance to splat plane along ray direction
				Vec3f y  = spos - origin;
				float yn = dot(y, snormal);			// if positive, splat is back-facing
				float dn = dot(direction, snormal);	// if positive, splat is back-facing
				float t = yn * (1.f / dn);
				if (t <= eps || t >= rayLen - eps) // epsilon based on ray length
					continue; // avoid accidental hits very close to origin and hitpoint

				// ray intersection on splat plane
				Vec3f p = origin + t * direction;
				float tpdist2 = (p - spos).lenSqr();
				if (tpdist2 > ssize * ssize)
					continue; // does not hit the splat

				// shrink it
				atomicMin(&((int*)in.radii)[i], __float_as_int(sqrtf(tpdist2)));
			}

			// Another leaf was postponed => process it as well.

			leafAddr = nodeAddr;
			if(nodeAddr < 0)
			{
				nodeAddr = *(int*)stackPtr;
				stackPtr -= 4;
			}
		} // leaf
	} // traversal
}

